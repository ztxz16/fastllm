#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>

#include "fastllm-cuda.cuh"
#include "fastllm.h"

// This will output the proper CUDA error strings in the event that a CUDA host
// call returns an error
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n", file, line,
            (int)err, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

static std::map<int, hipblasHandle_t> s_fastllmCublasHandleMap;
hipblasHandle_t getFastllmCublasHandle() {
    int id = -1;
    hipError_t err = hipGetDevice(&id);
    checkCudaErrors(err);

    auto it = s_fastllmCublasHandleMap.find(id);
    if (it != s_fastllmCublasHandleMap.end()) {
        return it->second;
    }
    hipblasHandle_t handler = nullptr;
    auto stat = hipblasCreate(&handler);

    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed:%d\n", stat);
        exit(0);
    } else {
        s_fastllmCublasHandleMap[id] = handler;
    }

    return handler;
}

#include <chrono>

double GetSpan(std::chrono::system_clock::time_point time1, std::chrono::system_clock::time_point time2) {
    auto duration = std::chrono::duration_cast<std::chrono::microseconds> (time2 - time1);
    return double(duration.count()) * std::chrono::microseconds::period::num / std::chrono::microseconds::period::den;
};

__global__ void FastllmCudaFloat2HalfKernel(float* a, half *b, int len) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len) {
        b[idx] = __float2half(a[idx]);
    }
}

__global__ void FastllmCudaInt82HalfKernel(uint8_t* a, float *scales, uint8_t *zeros, half *b, int len, int per) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len) {
        b[idx] = __float2half(scales[idx / per] * ((float)a[idx] - zeros[idx / per]));
    }
}

__global__ void FastllmCudaInt42HalfKernel(uint8_t* a, float *scales, float *mins, half *b, int len, int per) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len) {
        if (idx % 2 == 1) {
            b[idx] = __float2half(scales[idx / per] * (a[idx / 2] & 0xF) + mins[idx / per]);
        } else {
            b[idx] = __float2half(scales[idx / per] * (a[idx / 2] >> 4) + mins[idx / per]);
        }
    }
}

__global__ void FastllmCudaHalf2FlotaKernel(half* a, float *b, int len) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len) {
        b[idx] = __half2float(a[idx]);
    }
}

__global__ void FastllmCudaBiasKernel(float *a, float *bias, int k) {
    float *now = a + blockIdx.x * k;
    int stride = blockDim.x;
    for (int i = threadIdx.x; i < k; i += stride) {
        now[i] += bias[i];
    }
}

__global__ void FastllmGeluKernel(float* a, float *b, int len) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len) {
        float x = a[idx];
        b[idx] = 0.5f * x * (1.0f + tanhf(0.7978845608028654f * x * (1.0f + 0.044715f * x * x)));
    }
}

__global__ void FastllmSiluKernel(float* a, float *b, int len) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len) {
        float x = a[idx];
        b[idx] = x / (1.0 + expf(-x));
    }
}

__global__ void FastllmSwigluKernel(float* a, float *b, int len, int spatial, int mid) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len) {
        int id = idx / mid * spatial + idx % mid;
        float x = a[id], y = a[id + mid];
        b[idx] = (x / (1.0 + expf(-x))) * y;
    }
}

__global__ void FastllmMulKernel(float* a, float *b, float v, int len) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len) {
        b[idx] = a[idx] * v;
    }
}

__global__ void FastllmAddToKernel(float* a, float *b, float alpha, int len) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len) {
        a[idx] += b[idx] * alpha;
    }
}

__global__ void FastllmMulToKernel(float* a, float *b, float alpha, int len) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len) {
        a[idx] *= b[idx] * alpha;
    }
}

template <int THREAD_PER_BLOCK>
__global__ void FastllmAttentionMaskKernel(float* a, float *b, float maskValue, int n, int m, int spatial) {
    int on = blockIdx.x / m;
    int om = blockIdx.x % m;
    int o = on * m + om;
    int idx = threadIdx.x;
    for (int i = idx; i < spatial; i += THREAD_PER_BLOCK) {
        if (b[on * spatial + i] > 0.99) {
            a[o * spatial + i] = maskValue;
        }
    }
}

template <int THREAD_PER_BLOCK>
__global__ void FastllmAlibiMaskKernel(float* a, float *b, float maskValue, int n, int m, int spn, int spm, int spatial) {
    int on = blockIdx.x / m;
    int om = blockIdx.x % m;
    int o = on * m + om;
    int idx = threadIdx.x;
    float now = b[om];
    for (int i = idx; i < spatial; i += THREAD_PER_BLOCK) {
        int idi = i / spm, idj = i % spm;
        if (idj <= spm - spn + idi) {
            a[o * spatial + i] += now * idj;
        } else {
            a[o * spatial + i] = maskValue;
        }
    }
}

__global__ void FastllmPermuteKernel(float *dst, float *ori, int *temp, int axisLen, int len) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < len) {
        int old = 0;
        int idx = i;
        for (int j = 0; j < axisLen; ++j) {
            int order = temp[j];
            old += (idx / temp[j + 2 * axisLen]) * temp[order + 1 * axisLen];
            idx %= temp[j + 2 * axisLen];
        }
        dst[i] = ori[old];
    }
}

__global__ void FastllmLlamaRotatePosition2DKernel(float *data, float *positionIds, float *sin, float *cos,
                                                   int len, int bs, int spatial, int n, int m, int partStride, int sinCosStride, int rotateDim) {
    int o = (blockIdx.x / n);
    int l = o % len;
    int b = o / len;
    int j = threadIdx.x;
    int index = (int) (positionIds[b * partStride + l]);

    float curSin = sin[index * sinCosStride + j];
    float curCos = cos[index * sinCosStride + j];
    float *d = (float *) data + o * spatial + j;
    int i = blockIdx.x % n;
    float va = d[i * m], vb = d[i * m + m / 2];
    d[i * m] = va * curCos - vb * curSin;
    d[i * m + m / 2] = va * curSin + vb * curCos;
}

__global__ void FastllmNearlyRotatePosition2DKernel(float *data, float *positionIds, float *sin, float *cos,
                                                   int len, int bs, int spatial, int n, int m, int partStride, int sinCosStride, int rotateDim) {
/*
    int len = data.dims[0], bs = data.dims[1];
    int spatial = data.Count(2);
    int n = data.dims[2], m = data.dims[3];
    int stride = (int)sinData.dims[1];
    for (int l = 0; l < len; l++) {
        for (int b = 0; b < bs; b++) {
            int index = (int) ((float *) positionIds.cpuData)[(b * 2) * positionIds.dims.back() + l];
            float *sin = ((float*)sinData.cpuData) + stride * index;
            float *cos = ((float*)cosData.cpuData) + stride * index;
            float *d = (float *) data.cpuData + (l * bs + b) * spatial;
            for (int i = 0; i < n; i++) {
                int j = 0;
                for (; j < rotaryDim; j += 2) {
                    float a = d[j], b = d[j + 1];
                    d[j] = a * cos[j / 2] - b * sin[j / 2];
                    d[j + 1] = a * sin[j / 2] + b * cos[j / 2];
                }
                d += m;
            }
        }
    }
*/
    int o = (blockIdx.x / n);
    int l = o / bs;
    int b = o % bs;
    int j = threadIdx.x;
    int index = (int) (positionIds[b * 2 * partStride + l]);

    float curSin = sin[index * sinCosStride + j];
    float curCos = cos[index * sinCosStride + j];
    float *d = (float *) data + o * spatial + j * 2;
    int i = blockIdx.x % n;
    float va = d[i * m], vb = d[i * m + 1];
    d[i * m] = va * curCos - vb * curSin;
    d[i * m + 1] = va * curSin + vb * curCos;
}

__global__ void FastllmRotatePosition2DKernel(float *data, float *positionIds, float *sin, float *cos,
                                              int len, int bs, int spatial, int n, int m, int partStride, int sinCosStride, int rotateDim) {
    int o = (blockIdx.x / n) / 2;
    int l = o / bs;
    int b = o % bs;
    int part = (blockIdx.x / n) % 2;
    int j = threadIdx.x;
    int index = (int) (positionIds[(b * 2 + part) * partStride + l]);

    float curSin = sin[index * sinCosStride + j];
    float curCos = cos[index * sinCosStride + j];
    float *d = (float *) data + o * spatial + part * m / 2 + j;
    int i = blockIdx.x % n;
    float va = d[i * m], vb = d[i * m + m / 4];
    d[i * m] = va * curCos - vb * curSin;
    d[i * m + m / 4] = va * curSin + vb * curCos;
}

template <int THREAD_PER_BLOCK>
__global__ void FastllmSoftmaxKernelInner1(float* input, float *output, int outer, int channels) {
    int o = blockIdx.x;
    input = input + o * channels;
    output = output + o * channels;

    __shared__ float sdata[THREAD_PER_BLOCK];
    __shared__ float maxV;

    // 1. 每个线程计算一部分
    unsigned int tid = threadIdx.x;
    unsigned int per = (channels / THREAD_PER_BLOCK);
    unsigned int id = threadIdx.x * per;
    unsigned int len = per;
    if (tid == blockDim.x - 1) {
        len += (channels - per * THREAD_PER_BLOCK);
    }
    float maxValue = input[id];
    for (int i = 0; i < len; i++) {
        maxValue = max(maxValue, input[id + i]);
    }
    sdata[tid] = maxValue;
    __syncthreads();

    // 2. 求max
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = max(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }

    // 3. 记录max
    if (tid == 0) {
        maxV = sdata[0];
    }
    __syncthreads();

    // 4. 求和
    float sum = 0;
    for (int i = 0; i < len; i++) {
        output[id + i] = exp(input[id + i] - maxV);
        sum += output[id + i];
    }
    sdata[tid] = sum;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) {
        if (fabs(sdata[0]) < 1e-6) {
            sdata[0] = 0.1;
        }
    }
    __syncthreads();

    for (int i = 0; i < len; i++) {
        output[id + i] /= sdata[0];
    }
}


template <int THREAD_PER_BLOCK>
__global__ void FastllmRMSNormKernelInner1(float *input, float *weight, float *output, int outer, int channels, float eps) {
    int o = blockIdx.x;
    input = input + o * channels;
    output = output + o * channels;

    __shared__ float sdata2[THREAD_PER_BLOCK];
    __shared__ float scale;

    // 1. 每个线程计算一部分
    unsigned int tid = threadIdx.x;
    float sum2 = 0.0;
    for (int i = tid; i < channels; i += THREAD_PER_BLOCK) {
        float x = input[i];
        sum2 += x * x;
    }
    sdata2[tid] = sum2;
    __syncthreads();

    // 2. 求和
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata2[tid] += sdata2[tid + s];
        }
        __syncthreads();
    }

    // 3. 计算参数
    if (tid == 0) {
        scale = 1.0 / sqrt(sdata2[0] / channels + eps);
    }
    __syncthreads();

    for (int i = tid; i < channels; i += THREAD_PER_BLOCK) {
        output[i] = (input[i] * scale * weight[i]);
    }
}

template <int THREAD_PER_BLOCK>
__global__ void FastllmLayerNormKernelInner1(float *input, float *gamma, float *beta, float *output, int outer, int channels) {
    int o = blockIdx.x;
    input = input + o * channels;
    output = output + o * channels;

    __shared__ float sdata[THREAD_PER_BLOCK];
    __shared__ float sdata2[THREAD_PER_BLOCK];
    __shared__ float mean;
    __shared__ float var;

    // 1. 每个线程计算一部分
    unsigned int tid = threadIdx.x;
    float sum = 0.0, sum2 = 0.0;
    for (int i = tid; i < channels; i += THREAD_PER_BLOCK) {
        float x = input[i];
        sum += x;
        sum2 += x * x;
    }
    sdata[tid] = sum;
    sdata2[tid] = sum2;
    __syncthreads();

    // 2. 求和
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
            sdata2[tid] += sdata2[tid + s];
        }
        __syncthreads();
    }

    // 3. 计算参数
    if (tid == 0) {
        mean = sdata[0] / channels;
        var = sdata2[0] + mean * mean * channels - 2 * mean * channels * mean;
        var = sqrt(var / channels + 1e-10);
    }
    __syncthreads();

    for (int i = tid; i < channels; i += THREAD_PER_BLOCK) {
        output[i] = (input[i] - mean) / var * gamma[i] + beta[i];
    }
}

template <int THREAD_PER_BLOCK>
__global__ void FastllmLayerNormKernelTop1(float *input, float *output, int channels) {
    __shared__ float idData[THREAD_PER_BLOCK];
    __shared__ float maxData[THREAD_PER_BLOCK];
    float *inputData = input + blockIdx.x * channels;
    float *outputData = output + blockIdx.x * 2;
    int tid = threadIdx.x;
    maxData[tid] = -1e100;
    for (int j = tid; j < channels; j += THREAD_PER_BLOCK) {
        if (inputData[j] > maxData[tid]) {
            maxData[tid] = inputData[j];
            idData[tid] = j;
        }
    }
    __syncthreads();

    for (unsigned int s = THREAD_PER_BLOCK / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (maxData[tid] < maxData[tid + s]) {
                maxData[tid] = maxData[tid + s];
                idData[tid] = idData[tid + s];
            }
        }
        __syncthreads();
    }

    if (tid == 0) {
        outputData[0] = idData[0];
        outputData[1] = maxData[0];
    }
}

template <int NBlock, int MBlock, int KBlock>
__global__ void FastllmCudaBaseGemmKernelInt8(float *A, uint8_t *B, float *C,
                                              float *bias, float *scales, uint8_t *zeros,
                                              int n, int m, int k) {
    int nStart = blockIdx.x * NBlock, nEnd = nStart + NBlock;
    int kStart = blockIdx.y * KBlock, kEnd = kStart + KBlock;

    int id = kStart + threadIdx.x;
    __shared__ float shareA[NBlock * MBlock];
    __shared__ float shareB[KBlock * MBlock];
    float localSum[NBlock] = {0.0f};
    uint8_t zero = zeros[id];
    int idx = threadIdx.x >> 3;
    int idy = threadIdx.x & 7;
    for (int l = 0; l < m; l += MBlock) {
        if (threadIdx.x < MBlock) {
            for (int i = nStart; i < nEnd; i++) {
                if (i < n && l + threadIdx.x < m) {
                    shareA[(i - nStart) * MBlock + threadIdx.x] = A[i * m + l + threadIdx.x];
                } else {
                    shareA[(i - nStart) * MBlock + threadIdx.x] = 0.0f;
                }
            }
        }
        __syncthreads();
        if (threadIdx.x < MBlock) {
            for (int i = kStart; i < kEnd; i++) {
                if (i < k && l + threadIdx.x < m) {
                    shareB[(i - kStart) * MBlock + threadIdx.x] = B[i * m + l + threadIdx.x];
                } else {
                    shareB[(i - kStart) * MBlock + threadIdx.x] = 0.0f;
                }
            }
        }
        __syncthreads();

        for (int mStart = 0; mStart < MBlock; mStart += 4) {
            float curA[32] = {0.0f}, curB[32] = {0.0f};
            for (int i = 0; i < 8; i++) {
                for (int x = l + mStart; x < l + mStart + 4 && x < m; x++) {
                    curA[i * 4 + (x - l - mStart)] = shareA[(idx * 8 + i) * MBlock + (x - l)];
                }
            }
            for (int j = 0; j < 4; j++) {
                zero = zeros[kStart + (idy * 4 + j)];
                for (int x = l + mStart; x < l + mStart + 4 && x < m; x++) {
                    curB[j * 4 + (x - l - mStart)] = shareB[(idy * 4 + j) * MBlock + (x - l)] - zero;
                }
            }
            for (int i = 0; i < 8; i++) {
                for (int j = 0; j < 4; j++) {
                    int cur = i * 4 + j;
                    localSum[cur] += curA[i * 4 + 0] * curB[j * 4 + 0];
                    localSum[cur] += curA[i * 4 + 1] * curB[j * 4 + 1];
                    localSum[cur] += curA[i * 4 + 2] * curB[j * 4 + 2];
                    localSum[cur] += curA[i * 4 + 3] * curB[j * 4 + 3];
                }
            }
            __syncthreads();
        }
        __syncthreads();
    }

    for (int i = 0; i < 8; i++) {
        for (int j = 0; j < 4; j++) {
            if ((nStart + idx * 8 + i) < n && (kStart + idy * 4 + j) < k) {
                C[(nStart + idx * 8 + i) * k + (kStart + idy * 4 + j)] =
                        localSum[i * 4 + j] * scales[(kStart + idy * 4 + j)] + bias[(kStart + idy * 4 + j)];
            }
        }
    }
}

template <int THREAD_PER_BLOCK, int PART>
__global__ void FastllmGemvFp32Fp32Kernel2(float *A, float *B, float *C, float *bias, int m, int k) {
    __shared__ float sdata[THREAD_PER_BLOCK];
    unsigned int tid = threadIdx.x;

    // 1. 计算
    int st = blockIdx.x * PART;
    int end = st + PART;
    for (int p = st; p < end; p++) {
        sdata[tid] = 0;
        for (int i = tid; i < m; i += THREAD_PER_BLOCK) {
            sdata[tid] += A[i] * B[p * m + i];
        }
        __syncthreads();
        for (unsigned int s = 1; s < THREAD_PER_BLOCK; s *= 2) {
            if ((tid & (2 * s - 1)) == 0) {
                sdata[tid] += sdata[tid + s];
            }
            __syncthreads();
        }

        if (tid == 0) {
            C[p] = sdata[0] + bias[p];
        }
        __syncthreads();
    }
}

template <int THREAD_PER_BLOCK, int PART>
__global__ void FastllmGemvFp32Fp16Kernel2(float *A, half *B, float *C, float *bias, int m, int k) {
    __shared__ float sdata[THREAD_PER_BLOCK];
    unsigned int tid = threadIdx.x;

    // 1. 计算
    int st = blockIdx.x * PART;
    int end = st + PART;
    for (int p = st; p < end; p++) {
        sdata[tid] = 0;
        for (int i = tid; i < m; i += THREAD_PER_BLOCK) {
            sdata[tid] += A[i] * (float)B[p * m + i];
        }
        __syncthreads();
        for (unsigned int s = 1; s < THREAD_PER_BLOCK; s *= 2) {
            if ((tid & (2 * s - 1)) == 0) {
                sdata[tid] += sdata[tid + s];
            }
            __syncthreads();
        }

        if (tid == 0) {
            C[p] = sdata[0] + bias[p];
        }
        __syncthreads();
    }
}

template <int THREAD_PER_BLOCK, int PART>
__global__ void FastllmGemvInt8Kernel2(float *A, uint8_t *B, float *C,
                                       float *bias, float *scales, uint8_t *zeros,
                                       int m, int k) {
    __shared__ float sdata[THREAD_PER_BLOCK];
    unsigned int tid = threadIdx.x;

    // 1. 读入fdata
    /*for (int i = tid; i < m; i += THREAD_PER_BLOCK) {
        fdata[i] = A[i];
    }
    __syncthreads();*/

    // 2. 计算
    int st = blockIdx.x * PART;
    int end = st + PART;
    for (int p = st; p < end; p++) {
        sdata[tid] = 0;
        uint8_t zero = zeros[p];
        for (int i = tid; i < m; i += THREAD_PER_BLOCK) {
            sdata[tid] += A[i] * (B[p * m + i] - zero);
        }
        __syncthreads();
        for (unsigned int s = 1; s < THREAD_PER_BLOCK; s *= 2) {
            if ((tid & (2 * s - 1)) == 0) {
                sdata[tid] += sdata[tid + s];
            }
            __syncthreads();
        }

        if (tid == 0) {
            C[p] = sdata[0] * scales[p] + bias[p];
        }
        __syncthreads();
    }
}

template <int THREAD_PER_BLOCK, int SINGLE_COMPUTE, int REDUCE_NUMBER>
__global__ void FastllmGemvInt8Kernel1(float *A, uint8_t *B, float *C,
                                       float *bias, float *scales, uint8_t *zeros,
                                       int m, int k) {
    __shared__ float sdata[REDUCE_NUMBER];
    unsigned int tid = threadIdx.x;

    int part = m / REDUCE_NUMBER;
    // 1. 每个线程计算一部分
    for (int p = 0; p < part; p++) {
        float v[SINGLE_COMPUTE];
        for (int i = 0; i < SINGLE_COMPUTE; i++) {
            v[i] = A[p * REDUCE_NUMBER + tid * SINGLE_COMPUTE + i];
        }
        for (int i = 0; i < SINGLE_COMPUTE / part; i++) {
            float sum = 0;
            int colId = (blockIdx.x * SINGLE_COMPUTE / part + i);
            if (colId >= k) {
                sdata[i * (m / SINGLE_COMPUTE) + p * (REDUCE_NUMBER / SINGLE_COMPUTE) + tid] = 0;
                continue;
            }
            int id = colId * m + p * REDUCE_NUMBER + tid * SINGLE_COMPUTE;
            uint8_t zero = zeros[colId];
            for (int j = 0; j < SINGLE_COMPUTE; j++) {
                sum += v[j] * (B[id + j] - zero);
            }
            sdata[i * (m / SINGLE_COMPUTE) + p * (REDUCE_NUMBER / SINGLE_COMPUTE) + tid] = sum;
            __syncthreads();
        }
    }

    // 2. 求和
    for (unsigned int s = THREAD_PER_BLOCK / 2; s > 0; s >>= 1) {
        if (tid < s) {
            for (int i = 0; i < SINGLE_COMPUTE; i++) {
                sdata[i * THREAD_PER_BLOCK + tid] += sdata[i * THREAD_PER_BLOCK + tid + s];
            }
        }
        __syncthreads();
    }

    // 3. 写回结果
    if (tid == 0) {
        for (int i = 0; i < SINGLE_COMPUTE / part; i++) {
            int id = blockIdx.x * SINGLE_COMPUTE / part  + i;
            if (id >= k) {
                continue;
            }
            float sum = 0;
            for (int p = 0; p < part; p++) {
                sum += sdata[(i * part + p) * THREAD_PER_BLOCK];
            }
            C[id] = sum * scales[id] + bias[id];
        }
    }
}

template <int THREAD_PER_BLOCK, int PART>
__global__ void FastllmGemvInt4Kernel2(float *A, uint8_t *B, float *C,
                                       float *bias, float *scales, uint8_t *zeros,
                                       int m, int k) {
    __shared__ float sdata[THREAD_PER_BLOCK];
    unsigned int tid = threadIdx.x;

    // 1. 计算
    int st = blockIdx.x * PART;
    int end = st + PART;
    for (int p = st; p < end; p++) {
        sdata[tid] = 0;
        uint8_t zero = zeros[p];
        for (int i = tid; i < m / 2; i += THREAD_PER_BLOCK) {
            uint8_t now = B[p * m / 2 + i];
            sdata[tid] += (A[i * 2] * ((now >> 4) - zero) + A[i * 2 + 1] * ((now & 15) - zero));
        }
        __syncthreads();
        for (unsigned int s = 1; s < THREAD_PER_BLOCK; s *= 2) {
            if ((tid & (2 * s - 1)) == 0) {
                sdata[tid] += sdata[tid + s];
            }
            __syncthreads();
        }

        if (tid == 0) {
            C[p] = sdata[0] * scales[p] + bias[p];
        }
        __syncthreads();
    }
}

template <int THREAD_PER_BLOCK, int PART>
__global__ void FastllmGemvInt4NoZeroKernel2(float *A, uint8_t *B, float *C,
                                       float *bias, float *scales, float *mins,
                                       int m, int k) {
    __shared__ float sdata[THREAD_PER_BLOCK];
    unsigned int tid = threadIdx.x;

    // 1. 计算
    int st = blockIdx.x * PART;
    int end = st + PART;
    for (int p = st; p < end; p++) {
        sdata[tid] = 0;
        float minv = mins[p] / scales[p];
        for (int i = tid; i < m / 2; i += THREAD_PER_BLOCK) {
            uint8_t now = B[p * m / 2 + i];
            sdata[tid] += (A[i * 2] * (minv + (now >> 4)) + A[i * 2 + 1] * (minv + (now & 15)));
        }
        __syncthreads();
        for (unsigned int s = 1; s < THREAD_PER_BLOCK; s *= 2) {
            if ((tid & (2 * s - 1)) == 0) {
                sdata[tid] += sdata[tid + s];
            }
            __syncthreads();
        }

        if (tid == 0) {
            C[p] = sdata[0] * scales[p] + bias[p];
        }
        __syncthreads();
    }
}

void *FastllmCudaPrepareInput(const fastllm::Data &input) {
    void *ret;
    if (input.dataDevice == fastllm::DataDevice::CUDA) {
        ret = (void*)input.cudaData;
    } else {
        ret = (void*)FastllmCudaMalloc(input.expansionBytes);
        checkCudaErrors(hipMemcpy(ret, input.cpuData, input.expansionBytes, hipMemcpyHostToDevice));
    }
    return ret;
}

void FastllmCudaFinishInput(const fastllm::Data &input, void *data) {
    if (input.dataDevice != fastllm::DataDevice::CUDA) {
        FastllmCudaFree(data);
    }
}

void *FastllmCudaPrepareOutput(fastllm::Data &output) {
    void *ret;
    if (output.dataDevice == fastllm::DataDevice::CUDA) {
        ret = (float*)output.cudaData;
    } else {
        ret = (float*)FastllmCudaMalloc(output.expansionBytes);
    }
    return ret;
}

void FastllmCudaFinishOutput(fastllm::Data &output, void *data) {
    if (output.dataDevice != fastllm::DataDevice::CUDA) {
        checkCudaErrors(hipMemcpy(output.cpuData, data, output.expansionBytes, hipMemcpyDeviceToHost));
        FastllmCudaFree(data);
    }

    // hipDeviceSynchronize();
}

bool FastllmCudaMatMulFloatInt8(const fastllm::Data &input, fastllm::Data &weight, const fastllm::Data &bias, fastllm::Data &output, int n, int m, int k) {
    if (weight.cudaData == nullptr || weight.extraCudaData.size() == 0) {
        weight.ToDevice(fastllm::DataDevice::CUDA);

        float *cudaScales;
        checkCudaErrors(hipMalloc(&cudaScales, k * sizeof(float)));
        checkCudaErrors(hipMemcpy(cudaScales, weight.scales.data(), k * sizeof(float), hipMemcpyHostToDevice));
        weight.extraCudaData.push_back((void*)cudaScales);

        uint8_t *cudaZeropoints;
        checkCudaErrors(hipMalloc(&cudaZeropoints, k));
        uint8_t *zeropoints = new uint8_t[k];
        for (int i = 0; i < k; i++) {
            zeropoints[i] = weight.perChannelsConfigs[i].zeroPoint;
        }
        checkCudaErrors(hipMemcpy(cudaZeropoints, zeropoints, k, hipMemcpyHostToDevice));
        delete[] zeropoints;
        weight.extraCudaData.push_back((void*)cudaZeropoints);

        float *cudaBiasData;
        checkCudaErrors(hipMalloc(&cudaBiasData, k * sizeof(float)));
        if (bias.dims.size() > 0) {
            hipMemcpy(cudaBiasData, (uint8_t*)bias.cudaData, k * sizeof(float), hipMemcpyDeviceToDevice);
        } else {
            hipMemset(cudaBiasData, 0, k * sizeof(float));
        }
        weight.extraCudaData.push_back((void*)cudaBiasData);
    }

    float *cudaScales = (float*)weight.extraCudaData[0];
    uint8_t *cudaZeropoints = (uint8_t*)weight.extraCudaData[1];
    float *cudaBiasData = (float*)weight.extraCudaData[2];

    float *cudaInput = (float*)FastllmCudaPrepareInput(input);
    float *cudaOutput = (float*)FastllmCudaPrepareOutput(output);

    if (n >= 8) {
        auto fastllmCublasHandle = getFastllmCublasHandle();
        
        half *cudaFp16Input, *cudaFp16Output, *cudaFp16Weight;
        cudaFp16Input = (half *) FastllmCudaMalloc(n * m * sizeof(half));
        cudaFp16Output = (half *) FastllmCudaMalloc(n * k * sizeof(half));
        cudaFp16Weight = (half *) FastllmCudaMalloc(k * m * sizeof(half));

        __half h_alpha = __float2half_rn(1.0), h_beta = __float2half_rn(0.0);
        hipDataType AType = HIP_R_16F, BType = HIP_R_16F, CType = HIP_R_16F, ComputeType = HIP_R_16F;
        hipblasStatus_t status;

        int len = n * m;
        int threadPerBlock = min(256, len);
        FastllmCudaFloat2HalfKernel <<< (len - 1) / threadPerBlock + 1, threadPerBlock>>>(cudaInput, cudaFp16Input, len);

        len = k * m;
        FastllmCudaInt82HalfKernel <<< (len - 1) / threadPerBlock + 1, threadPerBlock>>>((uint8_t*)weight.cudaData,
                                                                                         cudaScales,
                                                                                         cudaZeropoints,
                                                                                         cudaFp16Weight, len, m);

        status = hipblasGemmEx(fastllmCublasHandle,
                              HIPBLAS_OP_T, HIPBLAS_OP_N,
                              k, n, m,
                              &h_alpha, cudaFp16Weight, AType,
                              m, cudaFp16Input, BType,
                              m, &h_beta,
                              cudaFp16Output, CType,
                              k, ComputeType, static_cast<hipblasGemmAlgo_t>(HIPBLAS_GEMM_DEFAULT));
        if (status != HIPBLAS_STATUS_SUCCESS) {
            printf("Error: cublas error.\n");
            exit(0);
        }

        len = n * k;
        FastllmCudaHalf2FlotaKernel <<< (len - 1) / threadPerBlock + 1, threadPerBlock >>>(cudaFp16Output, cudaOutput, len);
        FastllmCudaBiasKernel <<< n, 256 >>> (cudaOutput, cudaBiasData, k);

        FastllmCudaFree(cudaFp16Input);
        FastllmCudaFree(cudaFp16Output);
        FastllmCudaFree(cudaFp16Weight);
    } else {
        for (int i = 0; i < n; i++) {
            FastllmGemvInt8Kernel2<256, 1> <<< k, 256 >>>(cudaInput + i * m,
                                                          (uint8_t *) weight.cudaData,
                                                          cudaOutput + i * k,
                                                          cudaBiasData,
                                                          cudaScales,
                                                          cudaZeropoints,
                                                          m, k);
        }
    }
    FastllmCudaFinishInput(input, cudaInput);
    FastllmCudaFinishOutput(output, cudaOutput);
    return true;
}

bool FastllmCudaMatMulFloatInt4(const fastllm::Data &input, fastllm::Data &weight, const fastllm::Data &bias, fastllm::Data &output, int n, int m, int k) {
    if (weight.cudaData == nullptr || weight.extraCudaData.size() == 0) {
        weight.ToDevice(fastllm::DataDevice::CUDA);

        float *cudaScales;
        checkCudaErrors(hipMalloc(&cudaScales, k * sizeof(float)));
        checkCudaErrors(hipMemcpy(cudaScales, weight.scales.data(), k * sizeof(float), hipMemcpyHostToDevice));
        weight.extraCudaData.push_back((void*)cudaScales);

        uint8_t *cudaZeropoints;
        checkCudaErrors(hipMalloc(&cudaZeropoints, k));
        uint8_t *zeropoints = new uint8_t[k];
        for (int i = 0; i < k; i++) {
            zeropoints[i] = weight.perChannelsConfigs[i].zeroPoint;
        }
        checkCudaErrors(hipMemcpy(cudaZeropoints, zeropoints, k, hipMemcpyHostToDevice));
        delete[] zeropoints;
        weight.extraCudaData.push_back((void*)cudaZeropoints);

        float *cudaBiasData;
        checkCudaErrors(hipMalloc(&cudaBiasData, k * sizeof(float)));
        if (bias.dims.size() > 0) {
            checkCudaErrors(hipMemcpy(cudaBiasData, (uint8_t*)bias.cudaData, k * sizeof(float), hipMemcpyDeviceToDevice));
        } else {
            checkCudaErrors(hipMemset(cudaBiasData, 0, k * sizeof(float)));
        }
        weight.extraCudaData.push_back((void*)cudaBiasData);
    }

    float *cudaScales = (float*)weight.extraCudaData[0];
    uint8_t *cudaZeropoints = (uint8_t*)weight.extraCudaData[1];
    float *cudaBiasData = (float*)weight.extraCudaData[2];

    float *cudaInput = (float*)FastllmCudaPrepareInput(input);
    float *cudaOutput = (float*)FastllmCudaPrepareOutput(output);

    for (int i = 0; i < n; i++) {
        FastllmGemvInt4Kernel2<256, 1> <<< k, 256 >>>(cudaInput + i * m,
                                                      (uint8_t *) weight.cudaData,
                                                      cudaOutput + i * k,
                                                      cudaBiasData,
                                                      cudaScales,
                                                      cudaZeropoints,
                                                      m, k);
    }
    FastllmCudaFinishInput(input, cudaInput);
    FastllmCudaFinishOutput(output, cudaOutput);
    return true;
}

bool FastllmCudaMatMulFloatInt4NoZero(const fastllm::Data &input, fastllm::Data &weight, const fastllm::Data &bias, fastllm::Data &output, int n, int m, int k) {
    if (weight.cudaData == nullptr || weight.extraCudaData.size() == 0) {
        weight.ToDevice(fastllm::DataDevice::CUDA);

        float *cudaScales;
        checkCudaErrors(hipMalloc(&cudaScales, k * sizeof(float)));
        checkCudaErrors(hipMemcpy(cudaScales, weight.scales.data(), k * sizeof(float), hipMemcpyHostToDevice));
        weight.extraCudaData.push_back((void*)cudaScales);

        float *cudaMins;
        checkCudaErrors(hipMalloc(&cudaMins, k * sizeof(float)));
        float *mins = new float[k];
        for (int i = 0; i < k; i++) {
            mins[i] = weight.perChannelsConfigs[i].min;
        }
        checkCudaErrors(hipMemcpy(cudaMins, mins, k * sizeof(float), hipMemcpyHostToDevice));
        delete[] mins;
        weight.extraCudaData.push_back((void*)cudaMins);

        float *cudaBiasData;
        checkCudaErrors(hipMalloc(&cudaBiasData, k * sizeof(float)));
        if (bias.dims.size() > 0) {
            checkCudaErrors(hipMemcpy(cudaBiasData, (uint8_t*)bias.cudaData, k * sizeof(float), hipMemcpyDeviceToDevice));
        } else {
            checkCudaErrors(hipMemset(cudaBiasData, 0, k * sizeof(float)));
        }
        weight.extraCudaData.push_back((void*)cudaBiasData);
    }

    float *cudaScales = (float*)weight.extraCudaData[0];
    float *cudaMins = (float*)weight.extraCudaData[1];
    float *cudaBiasData = (float*)weight.extraCudaData[2];

    float *cudaInput = (float*)FastllmCudaPrepareInput(input);
    float *cudaOutput = (float*)FastllmCudaPrepareOutput(output);

    if (n >= 8) {
        auto fastllmCublasHandle = getFastllmCublasHandle();
        half *cudaFp16Input, *cudaFp16Output, *cudaFp16Weight;
        cudaFp16Input = (half *) FastllmCudaMalloc(n * m * sizeof(half));
        cudaFp16Output = (half *) FastllmCudaMalloc(n * k * sizeof(half));
        cudaFp16Weight = (half *) FastllmCudaMalloc(k * m * sizeof(half));

        __half h_alpha = __float2half_rn(1.0), h_beta = __float2half_rn(0.0);
        hipDataType AType = HIP_R_16F, BType = HIP_R_16F, CType = HIP_R_16F, ComputeType = HIP_R_16F;
        hipblasStatus_t status;

        int len = n * m;
        int threadPerBlock = min(256, len);
        FastllmCudaFloat2HalfKernel <<< (len - 1) / threadPerBlock + 1, threadPerBlock>>>(cudaInput, cudaFp16Input,
                                                                                          len);

        len = k * m;
        FastllmCudaInt42HalfKernel <<< (len - 1) / threadPerBlock + 1, threadPerBlock>>>((uint8_t *) weight.cudaData,
                                                                                         cudaScales,
                                                                                         cudaMins,
                                                                                         cudaFp16Weight, len, m);

        status = hipblasGemmEx(fastllmCublasHandle,
                              HIPBLAS_OP_T, HIPBLAS_OP_N,
                              k, n, m,
                              &h_alpha, cudaFp16Weight, AType,
                              m, cudaFp16Input, BType,
                              m, &h_beta,
                              cudaFp16Output, CType,
                              k, ComputeType, static_cast<hipblasGemmAlgo_t>(HIPBLAS_GEMM_DEFAULT));
        if (status != HIPBLAS_STATUS_SUCCESS) {
            printf("Error: cublas error.\n");
            exit(0);
        }

        len = n * k;
        FastllmCudaHalf2FlotaKernel <<< (len - 1) / threadPerBlock + 1, threadPerBlock >>>(cudaFp16Output, cudaOutput,
                                                                                           len);
        FastllmCudaBiasKernel <<< n, 256 >>>(cudaOutput, cudaBiasData, k);

        FastllmCudaFree(cudaFp16Input);
        FastllmCudaFree(cudaFp16Output);
        FastllmCudaFree(cudaFp16Weight);
    } else {
    for (int i = 0; i < n; i++) {
        FastllmGemvInt4NoZeroKernel2<256, 1> <<< k, 256 >>>(cudaInput + i * m,
                                                      (uint8_t *) weight.cudaData,
                                                      cudaOutput + i * k,
                                                      cudaBiasData,
                                                      cudaScales,
                                                      cudaMins,
                                                      m, k);
    }
    }
    FastllmCudaFinishInput(input, cudaInput);
    FastllmCudaFinishOutput(output, cudaOutput);
    return true;
}

bool FastllmCudaMatMulFloat32(const fastllm::Data &input, fastllm::Data &weight, const fastllm::Data &bias, fastllm::Data &output, int n, int m, int k) {
    if (weight.cudaData == nullptr || weight.extraCudaData.size() == 0) {
        weight.ToDevice(fastllm::DataDevice::CUDA);
        float *cudaBiasData;
        checkCudaErrors(hipMalloc(&cudaBiasData, k * sizeof(float)));
        if (bias.dims.size() > 0) {
            checkCudaErrors(hipMemcpy(cudaBiasData, (uint8_t*)bias.cudaData, k * sizeof(float), hipMemcpyDeviceToDevice));
        } else {
            checkCudaErrors(hipMemset(cudaBiasData, 0, k * sizeof(float)));
        }
        weight.extraCudaData.push_back((void*)cudaBiasData);
    }

    float *cudaBiasData = (float*)weight.extraCudaData[0];
    float *cudaInput = (float*)FastllmCudaPrepareInput(input);
    float *cudaOutput = (float*)FastllmCudaPrepareOutput(output);

    if (n > 1) {
        float h_alpha = 1.0, h_beta = 0.0;
        auto fastllmCublasHandle = getFastllmCublasHandle();
        //hipDeviceSynchronize();
        hipDataType AType = HIP_R_32F, BType = HIP_R_32F, CType = HIP_R_32F, ComputeType = HIP_R_32F;
        hipblasStatus_t status;

        status = hipblasGemmEx(fastllmCublasHandle,
                              HIPBLAS_OP_T, HIPBLAS_OP_N,
                              k, n, m,
                              &h_alpha, weight.cudaData, AType,
                              m, cudaInput, BType,
                              m, &h_beta,
                              cudaOutput, CType,
                              k, ComputeType, static_cast<hipblasGemmAlgo_t>(HIPBLAS_GEMM_DEFAULT));
        if (status != HIPBLAS_STATUS_SUCCESS) {
            printf("Error: cublas error.\n");
            FastllmCudaFinishInput(input, cudaInput);
            FastllmCudaFinishOutput(output, cudaOutput);
            exit(0);
        }

        FastllmCudaBiasKernel <<< n, 256 >>> (cudaOutput, (float*)weight.extraCudaData[0], k);
    } else {
        FastllmGemvFp32Fp32Kernel2<256, 1> <<< k, 256 >>>(cudaInput, (float *) weight.cudaData, cudaOutput, cudaBiasData, m, k);
    }

    FastllmCudaFinishInput(input, cudaInput);
    FastllmCudaFinishOutput(output, cudaOutput);
    return true;
}

bool FastllmCudaMatMulFloat16(const fastllm::Data &input, fastllm::Data &weight, const fastllm::Data &bias, fastllm::Data &output, int n, int m, int k) {
    if (weight.cudaData == nullptr || weight.extraCudaData.size() == 0) {
        weight.ToDevice(fastllm::DataDevice::CUDA);
        float *cudaBiasData;
        checkCudaErrors(hipMalloc(&cudaBiasData, k * sizeof(float)));
        if (bias.dims.size() > 0) {
            checkCudaErrors(hipMemcpy(cudaBiasData, (uint8_t*)bias.cudaData, k * sizeof(float), hipMemcpyDeviceToDevice));
        } else {
            checkCudaErrors(hipMemset(cudaBiasData, 0, k * sizeof(float)));
        }
        weight.extraCudaData.push_back((void*)cudaBiasData);
    }
    float *cudaBiasData = (float*)weight.extraCudaData[0];
    float *cudaInput = (float*)FastllmCudaPrepareInput(input);
    float *cudaOutput = (float*)FastllmCudaPrepareOutput(output);

    if (n > 1) {
        half *cudaFp16Input, *cudaFp16Output;
        cudaFp16Input = (half *) FastllmCudaMalloc(n * m * sizeof(half));
        cudaFp16Output = (half *) FastllmCudaMalloc(n * k * sizeof(half));

        __half h_alpha = __float2half_rn(1.0), h_beta = __float2half_rn(0.0);
        auto fastllmCublasHandle = getFastllmCublasHandle();
        //hipDeviceSynchronize();
        hipDataType AType = HIP_R_16F, BType = HIP_R_16F, CType = HIP_R_16F, ComputeType = HIP_R_16F;
        hipblasStatus_t status;

        int len = n * m;
        int threadPerBlock = min(256, len);
        FastllmCudaFloat2HalfKernel <<< (len - 1) / threadPerBlock + 1, threadPerBlock>>>(cudaInput, cudaFp16Input,
                                                                                          len);

        status = hipblasGemmEx(fastllmCublasHandle,
                              HIPBLAS_OP_T, HIPBLAS_OP_N,
                              k, n, m,
                              &h_alpha, (half *) weight.cudaData, AType,
                              m, cudaFp16Input, BType,
                              m, &h_beta,
                              cudaFp16Output, CType,
                              k, ComputeType, static_cast<hipblasGemmAlgo_t>(HIPBLAS_GEMM_DEFAULT));
        if (status != HIPBLAS_STATUS_SUCCESS) {
            printf("Error: cublas error.\n");
            exit(0);
        }

        len = n * k;
        FastllmCudaHalf2FlotaKernel <<< (len - 1) / threadPerBlock + 1, threadPerBlock >>>(cudaFp16Output, cudaOutput,
                                                                                           len);
        FastllmCudaBiasKernel <<< n, 256 >>> (cudaOutput, (float*)weight.extraCudaData[0], k);
        //hipDeviceSynchronize();

        FastllmCudaFree(cudaFp16Input);
        FastllmCudaFree(cudaFp16Output);
    } else {
        FastllmGemvFp32Fp16Kernel2<256, 1> <<< k, 256 >>>(cudaInput, (half *) weight.cudaData, cudaOutput, cudaBiasData, m, k);
    }

    FastllmCudaFinishInput(input, cudaInput);
    FastllmCudaFinishOutput(output, cudaOutput);
    return true;
}

struct CudaMemoryBuffer {
    void *data;
    size_t size;
    bool busy;

    CudaMemoryBuffer () {}

    CudaMemoryBuffer (void *data, size_t size, bool busy) :
            data(data), size(size), busy(busy) {}
};
std::map<int, std::vector <CudaMemoryBuffer>> cudaBuffersMap;
std::map<int, std::vector <CudaMemoryBuffer>> bigBuffersMap;

void * FastllmCudaMalloc(size_t size) {
    int id = -1;
    hipError_t err = hipGetDevice(&id);
    checkCudaErrors(err);

    if (size > 1024 * 1024) {
        auto bigBuffers = bigBuffersMap[id];
        int selId = -1;
        for (int i = 0; i < bigBuffers.size(); i++) {
            if (bigBuffers[i].size >= size && !bigBuffers[i].busy
                && bigBuffers[i].size - size < 32 * 1024 * 1024) {
                if (selId == -1 || bigBuffers[selId].size > bigBuffers[i].size) {
                    selId = i;
                }
            }
        }
        if (selId != -1) {
            bigBuffers[selId].busy = true;
            return bigBuffers[selId].data;
        }

        void * ret;
        checkCudaErrors(hipMalloc(&ret, size));
        bigBuffers.push_back(CudaMemoryBuffer(ret, size, true));
        return ret;
    }
    auto cudaBuffers = cudaBuffersMap[id];
    for (int i = 0; i < cudaBuffers.size(); i++) {
        if (cudaBuffers[i].size >= size && !cudaBuffers[i].busy) {
            cudaBuffers[i].busy = true;
            return cudaBuffers[i].data;
        }
    }
    void * ret;
    checkCudaErrors(hipMalloc(&ret, size));
    cudaBuffers.push_back(CudaMemoryBuffer(ret, size, true));
    return ret;
}

void FastllmCudaFree(void *ret) {
    int id = -1;
    hipError_t err = hipGetDevice(&id);
    checkCudaErrors(err);

    auto cudaBuffers = cudaBuffersMap[id];
    for (int i = 0; i < cudaBuffers.size(); i++) {
        if (cudaBuffers[i].data == ret) {
            cudaBuffers[i].busy = false;
            return;
}
    }
    auto bigBuffers = bigBuffersMap[id];
    for (int i = 0; i < bigBuffers.size(); i++) {
        if (bigBuffers[i].data == ret) {
            bigBuffers[i].busy = false;
            return;
        }
    }
    checkCudaErrors(hipFree(ret));
}

void FastllmCudaMallocBigBuffer(size_t size) {
    void * ret;
    int id = -1;
    checkCudaErrors(hipMalloc(&ret, size));
    auto bigBuffers = bigBuffersMap[id];
    bigBuffers.push_back(CudaMemoryBuffer(ret, size, false));
}

void FastllmCudaClearBigBuffer() {
    int id = -1;
    auto bigBuffers = bigBuffersMap[id];

    std::vector <CudaMemoryBuffer> temp;
    for (int i = 0; i < bigBuffers.size(); i++) {
        if (!bigBuffers[i].busy) {
            checkCudaErrors(hipFree(bigBuffers[i].data));
        } else {
            temp.push_back(bigBuffers[i]);
        }
    }
    bigBuffers.clear();
    bigBuffers = temp;
}

void FastllmCudaCopyFromHostToDevice(void *dst, void *src, size_t size) {
    checkCudaErrors(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
    //hipDeviceSynchronize();
}

void FastllmCudaCopyFromDeviceToHost(void *dst, void *src, size_t size) {
    checkCudaErrors(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost));
    //hipDeviceSynchronize();
}

void FastllmCudaCopyFromDeviceToDevice(void *dst, void *src, size_t size) {
    checkCudaErrors(hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice));
    //hipDeviceSynchronize();
}

void FastllmCudaMemcpy2DDeviceToDevice(void * 	dst, size_t 	dpitch, const void * 	src,
                                       size_t 	spitch, size_t 	width, size_t 	height) {
    checkCudaErrors(hipMemcpy2D(dst, dpitch, src, spitch, width, height, hipMemcpyDeviceToDevice));
    // hipDeviceSynchronize();
}

bool FastllmCudaGeluNew(const fastllm::Data &input, fastllm::Data &output) {
    int len = input.Count(0);
    float *cudaInput = (float *) FastllmCudaPrepareInput(input);
    float *cudaOutput = (float *) FastllmCudaPrepareOutput(output);
    int threadPerBlock = min(256, len);
    FastllmGeluKernel <<< (len - 1) / threadPerBlock + 1, threadPerBlock>>>(cudaInput, cudaOutput, len);
    FastllmCudaFinishInput(input, cudaInput);
    FastllmCudaFinishOutput(output, cudaOutput);
    return true;
}

bool FastllmCudaSilu(const fastllm::Data &input, fastllm::Data &output) {
    int len = input.Count(0);
    float *cudaInput = (float *) FastllmCudaPrepareInput(input);
    float *cudaOutput = (float *) FastllmCudaPrepareOutput(output);
    int threadPerBlock = min(256, len);
    FastllmSiluKernel <<< (len - 1) / threadPerBlock + 1, threadPerBlock>>>(cudaInput, cudaOutput, len);
    FastllmCudaFinishInput(input, cudaInput);
    FastllmCudaFinishOutput(output, cudaOutput);
    return true;
}

bool FastllmCudaSwiglu(const fastllm::Data &input, fastllm::Data &output) {
    int len = output.Count(0);
    float *cudaInput = (float *) FastllmCudaPrepareInput(input);
    float *cudaOutput = (float *) FastllmCudaPrepareOutput(output);
    int spatial = input.Count(input.dims.size() - 1), mid = spatial / 2;

    int threadPerBlock = min(256, len);
    FastllmSwigluKernel <<< (len - 1) / threadPerBlock + 1, threadPerBlock>>>(cudaInput, cudaOutput, len, spatial, mid);

    FastllmCudaFinishInput(input, cudaInput);
    FastllmCudaFinishOutput(output, cudaOutput);
    return true;
}

bool FastllmCudaMul(const fastllm::Data &input, float v, fastllm::Data &output) {
    int len = input.Count(0);
    float *cudaInput = (float *) FastllmCudaPrepareInput(input);
    float *cudaOutput = (float *) FastllmCudaPrepareOutput(output);
    int threadPerBlock = min(256, len);
    FastllmMulKernel <<< (len - 1) / threadPerBlock + 1, threadPerBlock>>>(cudaInput, cudaOutput, v, len);
    FastllmCudaFinishInput(input, cudaInput);
    FastllmCudaFinishOutput(output, cudaOutput);
    return true;
}

bool FastllmCudaAddTo(fastllm::Data &input0, const fastllm::Data &input1, float alpha) {
    int len = input0.Count(0);
    float *cudaData = (float *) FastllmCudaPrepareInput(input0);
    float *input1Data = (float *) FastllmCudaPrepareInput(input1);

    int threadPerBlock = min(256, len);
    FastllmAddToKernel <<< (len - 1) / threadPerBlock + 1, threadPerBlock>>>(cudaData, input1Data, alpha, len);
    FastllmCudaFinishInput(input1, input1Data);
    FastllmCudaFinishOutput(input0, cudaData);
    return true;
}

bool FastllmCudaMulTo(fastllm::Data &input0, const fastllm::Data &input1, float alpha) {
    int len = input0.Count(0);
    float *cudaData = (float *) FastllmCudaPrepareInput(input0);
    float *input1Data = (float *) FastllmCudaPrepareInput(input1);

    int threadPerBlock = min(256, len);
    FastllmMulToKernel <<< (len - 1) / threadPerBlock + 1, threadPerBlock>>>(cudaData, input1Data, alpha, len);
    FastllmCudaFinishInput(input1, input1Data);
    FastllmCudaFinishOutput(input0, cudaData);
    return true;
}

bool FastllmCudaAttentionMask(fastllm::Data &input, const fastllm::Data &mask, float maskValue) {
    int spatial = input.Count(2), n = input.dims[0], m = input.dims[1];
    float *cudaData = (float *) FastllmCudaPrepareInput(input);
    float *maskData = (float *) FastllmCudaPrepareInput(mask);

    FastllmAttentionMaskKernel <256> <<< n * m, 256>>>(cudaData, maskData, maskValue,
                                                       n, m, spatial);
    FastllmCudaFinishInput(mask, maskData);
    FastllmCudaFinishOutput(input, cudaData);
    return true;
}

bool FastllmCudaAlibiMask(fastllm::Data &input, const fastllm::Data &mask, float maskValue) {
    int n = input.dims[0], m = input.dims[1];
    int spn = input.dims[2], spm = input.dims[3];
    int spatial = input.Count(2);
    float *cudaData = (float *) FastllmCudaPrepareInput(input);
    float *maskData = (float *) FastllmCudaPrepareInput(mask);

    FastllmAlibiMaskKernel <256> <<< n * m, 256>>>(cudaData, maskData, maskValue,
                                                   n, m, spn, spm, spatial);
    FastllmCudaFinishInput(mask, maskData);
    FastllmCudaFinishOutput(input, cudaData);
    return true;
}

bool FastllmCudaSoftmax(const fastllm::Data &input, fastllm::Data &output, int axis) {
    float *cudaInput = (float *) FastllmCudaPrepareInput(input);
    float *cudaOutput = (float *) FastllmCudaPrepareInput(output);

    int dimsLen = input.dims.size();
    axis = (axis % dimsLen + dimsLen) % dimsLen;
    int outer = input.Count(0) / input.Count(axis);
    int channels = input.dims[axis];
    int inner = input.Count(axis + 1);

    if (inner == 1) {
        if (channels < 8) {
            FastllmSoftmaxKernelInner1 <1> <<< outer, 1 >>> (cudaInput, cudaOutput, outer, channels);
        } else if (channels < 64) {
            FastllmSoftmaxKernelInner1 <8> <<< outer, 8 >>> (cudaInput, cudaOutput, outer, channels);
        } else if (channels < 512) {
            FastllmSoftmaxKernelInner1 <64> <<< outer, 64 >>> (cudaInput, cudaOutput, outer, channels);
        } else {
            FastllmSoftmaxKernelInner1 <256> <<< outer, 256 >>> (cudaInput, cudaOutput, outer, channels);
        }

    } else {
        printf("softmax error.\n");
        exit(0);
    }

    FastllmCudaFinishInput(input, cudaInput);
    FastllmCudaFinishOutput(output, cudaOutput);
    return true;
}

bool FastllmCudaRMSNorm(const fastllm::Data &input, fastllm::Data &weight, fastllm::Data &output, float eps) {
    weight.ToDevice(fastllm::DataDevice::CUDA);

    float *cudaInput = (float *) FastllmCudaPrepareInput(input);
    float *cudaOutput = (float *) FastllmCudaPrepareInput(output);

    int dimsLen = input.dims.size();
    int axis = dimsLen - 1;
    int outer = input.Count(0) / input.Count(axis);
    int channels = input.dims[axis];

    if (channels < 64) {
        FastllmRMSNormKernelInner1<1> <<< outer, 1 >>>(cudaInput, (float *) weight.cudaData, cudaOutput, outer, channels, eps);
    } else if (channels < 512) {
        FastllmRMSNormKernelInner1<64> <<< outer, 64 >>>(cudaInput, (float *) weight.cudaData, cudaOutput, outer, channels, eps);
    } else {
        FastllmRMSNormKernelInner1<512> <<< outer, 512 >>>(cudaInput, (float *) weight.cudaData, cudaOutput, outer, channels, eps);
    }

    FastllmCudaFinishInput(input, cudaInput);
    FastllmCudaFinishOutput(output, cudaOutput);
    return true;
}

bool FastllmCudaLayerNorm(const fastllm::Data &input, fastllm::Data &gamma, fastllm::Data &beta, fastllm::Data &output, int axis) {
    gamma.ToDevice(fastllm::DataDevice::CUDA);
    beta.ToDevice(fastllm::DataDevice::CUDA);

    float *cudaInput = (float *) FastllmCudaPrepareInput(input);
    float *cudaOutput = (float *) FastllmCudaPrepareInput(output);

    int dimsLen = input.dims.size();
    axis = (axis % dimsLen + dimsLen) % dimsLen;
    int outer = input.Count(0) / input.Count(axis);
    int channels = input.dims[axis];
    int inner = input.strides[axis];

    if (inner == 1) {
        if (channels < 64) {
            FastllmLayerNormKernelInner1<1> <<< outer, 1 >>>(cudaInput, (float *) gamma.cudaData,
                                                             (float *) beta.cudaData, cudaOutput,
                                                             outer, channels);
        } else if (channels < 512) {
            FastllmLayerNormKernelInner1<64> <<< outer, 64 >>>(cudaInput, (float *) gamma.cudaData,
                                                               (float *) beta.cudaData, cudaOutput,
                                                               outer, channels);
        } else {
            FastllmLayerNormKernelInner1<512> <<< outer, 512 >>>(cudaInput, (float *) gamma.cudaData,
                                                                 (float *) beta.cudaData, cudaOutput,
                                                                 outer, channels);
        }
    } else {
        printf("layernorm error.\n");
        exit(0);
    }

    FastllmCudaFinishInput(input, cudaInput);
    FastllmCudaFinishOutput(output, cudaOutput);
    return true;
}

bool FastllmCudaTopK(const fastllm::Data &input, fastllm::Data &output, int topk) {
    if (topk != 1) {
        printf("topk: unsupport topk > 1.");
        exit(0);
    }

    float *cudaInput = (float *) FastllmCudaPrepareInput(input);
    float *cudaOutput = (float *) FastllmCudaPrepareInput(output);

    int dimsLen = input.dims.size();
    int outer = input.Count(0) / input.Count(dimsLen - 1);
    int channels = input.dims[dimsLen - 1];

    FastllmLayerNormKernelTop1 <256> <<< outer, 256 >>> (cudaInput, cudaOutput, channels);
    FastllmCudaFinishInput(input, cudaInput);
    FastllmCudaFinishOutput(output, cudaOutput);
    return true;
}

bool FastllmCudaPermute(fastllm::Data &input, const std::vector<int> &axis) {
    if (input.dataDevice != fastllm::DataDevice::CUDA) {
        printf("permute: data should in cuda.\n");
        exit(0);
    }
    int len = input.Count(0);
    float *tempData = (float *)FastllmCudaMalloc(len * sizeof(float));
    checkCudaErrors(hipMemcpy(tempData, input.cudaData, len * sizeof(float), hipMemcpyDeviceToDevice));

    std::vector<int> new_dims;
    for (int i = 0; i < axis.size(); i++) {
        new_dims.push_back(input.dims[axis[i]]);
    }

    {
        std::vector<int> temp;
        int len = input.Count(0);
        for (int i = 0; i < axis.size(); i++) {
            temp.push_back(axis[i]);
        }
        for (int i = 0; i < axis.size(); i++) {
            temp.push_back(input.Count(i + 1));
        }
        input.Resize(new_dims);
        for (int i = 0; i < axis.size(); i++) {
            temp.push_back(input.Count(i + 1));
        }

        int *cudaTemp = (int*)FastllmCudaMalloc(temp.size() * sizeof(int));
        checkCudaErrors(hipMemcpy(cudaTemp, temp.data(), temp.size() * sizeof(int), hipMemcpyHostToDevice));
        int threadPerBlock = min(256, len);
        FastllmPermuteKernel <<< (len - 1) / threadPerBlock + 1, threadPerBlock >>> ((float*)input.cudaData, tempData, cudaTemp, (int)axis.size(), len);
        FastllmCudaFree(cudaTemp);
    }

    FastllmCudaFree(tempData);
    return true;
}

bool FastllmCudaBatchMatMul(const fastllm::Data &input0, const fastllm::Data &input1, fastllm::Data &output,
                            int input0Spatial, int input1Spatial, int outputSpatial,
                            int input0Stride, int input1Stride,
                            int batch, int n, int m, int k, float alpha) {
    float *cudaInput0 = (float *) FastllmCudaPrepareInput(input0);
    float *cudaInput1 = (float *) FastllmCudaPrepareInput(input1);
    float *cudaOutput = (float *) FastllmCudaPrepareOutput(output);
    float beta = 0;

    auto fastllmCublasHandle = getFastllmCublasHandle();
    hipblasStatus_t status;

    status = hipblasSgemmStridedBatched(fastllmCublasHandle,
                                       HIPBLAS_OP_N, HIPBLAS_OP_N,
                                       k, n, m, &alpha,
                                       cudaInput1, input1Stride, input1Spatial,
                                       cudaInput0, input0Stride, input0Spatial,
                                       &beta,
                                       cudaOutput, k, k * n, batch);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("status = %d\n", (int)status);
        printf("%d %d %d\n", k, n, m);
        printf("Error: cublas error.\n");
        throw("cublas error");
        exit(0);
    }

    FastllmCudaFinishInput(input0, cudaInput0);
    FastllmCudaFinishInput(input1, cudaInput1);
    FastllmCudaFinishOutput(output, cudaOutput);
    return true;
}

bool FastllmCudaBatchMatMulTransB(const fastllm::Data &input0, const fastllm::Data &input1, fastllm::Data &output,
                                  int input0Spatial, int input1Spatial, int outputSpatial,
                                  int input0Stride, int input1Stride,
                                  int batch, int n, int m, int k, float alpha) {
    float *cudaInput0 = (float *) FastllmCudaPrepareInput(input0);
    float *cudaInput1 = (float *) FastllmCudaPrepareInput(input1);
    float *cudaOutput = (float *) FastllmCudaPrepareOutput(output);
    float beta = 0;
    auto fastllmCublasHandle = getFastllmCublasHandle();
    hipblasStatus_t status;

    status = hipblasSgemmStridedBatched(fastllmCublasHandle,
                                       HIPBLAS_OP_T, HIPBLAS_OP_N,
                                       k, n, m, &alpha,
                                       cudaInput1, input1Stride, input1Spatial,
                                       cudaInput0, input0Stride, input0Spatial,
                                       &beta,
                                       cudaOutput, k, k * n, batch);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("status = %d\n", (int)status);
        printf("%d %d %d\n", k, n, m);
        printf("Error: cublas error.\n");
        throw("cublas error");
        exit(0);
    }

    FastllmCudaFinishInput(input0, cudaInput0);
    FastllmCudaFinishInput(input1, cudaInput1);
    FastllmCudaFinishOutput(output, cudaOutput);
    return true;
}

bool FastllmCudaRotatePosition2D(fastllm::Data &data, const fastllm::Data &positionIds,
                                 const fastllm::Data &sinData, const fastllm::Data &cosData, int rotaryDim) {
    float *cudaData = (float *) FastllmCudaPrepareInput(data);
    float *cudaPositionIds = (float *) FastllmCudaPrepareInput(positionIds);
    float *cudaSin = (float *) FastllmCudaPrepareInput(sinData);
    float *cudaCos = (float *) FastllmCudaPrepareInput(cosData);

    int outer = data.dims[0] * data.dims[1];
    int spatial = data.Count(2);
    int len = data.dims[0], bs = data.dims[1];
    int n = data.dims[2], m = data.dims[3];
    FastllmRotatePosition2DKernel <<< outer * 2 * n, min(rotaryDim, m / 4) >>> (cudaData, cudaPositionIds, cudaSin, cudaCos,
                                                                                len, bs, spatial, n, m,
                                                                                (int)positionIds.dims.back(), (int)sinData.dims[1], rotaryDim);

    FastllmCudaFinishInput(positionIds, cudaPositionIds);
    FastllmCudaFinishInput(sinData, cudaSin);
    FastllmCudaFinishInput(cosData, cudaCos);
    FastllmCudaFinishOutput(data, cudaData);

    return true;
}

bool FastllmCudaNearlyRotatePosition2D(fastllm::Data &data, const fastllm::Data &positionIds,
                                 const fastllm::Data &sinData, const fastllm::Data &cosData, int rotaryDim) {
    float *cudaData = (float *) FastllmCudaPrepareInput(data);
    float *cudaPositionIds = (float *) FastllmCudaPrepareInput(positionIds);
    float *cudaSin = (float *) FastllmCudaPrepareInput(sinData);
    float *cudaCos = (float *) FastllmCudaPrepareInput(cosData);

    int outer = data.dims[0] * data.dims[1];
    int spatial = data.Count(2);
    int len = data.dims[0], bs = data.dims[1];
    int n = data.dims[2], m = data.dims[3];
    FastllmNearlyRotatePosition2DKernel <<< outer * n, min(rotaryDim, m / 4) >>> (cudaData, cudaPositionIds, cudaSin, cudaCos,
                                                                                len, bs, spatial, n, m,
                                                                                (int)positionIds.dims.back(), (int)sinData.dims[1], rotaryDim);

    FastllmCudaFinishInput(positionIds, cudaPositionIds);
    FastllmCudaFinishInput(sinData, cudaSin);
    FastllmCudaFinishInput(cosData, cudaCos);
    FastllmCudaFinishOutput(data, cudaData);
    return true;
}

bool FastllmCudaLlamaRotatePosition2D(fastllm::Data &data, const fastllm::Data &positionIds,
                                      const fastllm::Data &sinData, const fastllm::Data &cosData, int rotaryDim) {
    float *cudaData = (float *) FastllmCudaPrepareInput(data);
    float *cudaPositionIds = (float *) FastllmCudaPrepareInput(positionIds);
    float *cudaSin = (float *) FastllmCudaPrepareInput(sinData);
    float *cudaCos = (float *) FastllmCudaPrepareInput(cosData);

    int outer = data.dims[0] * data.dims[1];
    int spatial = data.Count(2);
    int bs = data.dims[0], len = data.dims[1];
    int n = data.dims[2], m = data.dims[3];
    FastllmLlamaRotatePosition2DKernel <<< outer * n, min(rotaryDim, m / 2) >>> (cudaData, cudaPositionIds, cudaSin, cudaCos,
                                                                                 len, bs, spatial, n, m,
                                                                                 (int)positionIds.dims.back(), (int)sinData.dims[1], rotaryDim);

    FastllmCudaFinishInput(positionIds, cudaPositionIds);
    FastllmCudaFinishInput(sinData, cudaSin);
    FastllmCudaFinishInput(cosData, cudaCos);
    FastllmCudaFinishOutput(data, cudaData);
    return true;
}

void FastllmCudaSetDevice(int gpu_id) {
    hipSetDevice(gpu_id);
}
